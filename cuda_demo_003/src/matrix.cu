#include "hip/hip_runtime.h"
#include "matrix.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include ""

#define THREAD_NUM 256
#define MATRIX_SIZE 2000

const int blocks_num = MATRIX_SIZE * (MATRIX_SIZE + THREAD_NUM - 1);

bool initCUDA() {
    int count;
    hipGetDeviceCount(&count);

    if (count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    int i;

    for(i = 0; i< count; i++) {
        hipDeviceProp_t prop;

        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1) {
                break;
            }
        }
    }
    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    hipSetDevice(i);

    return true;
}

__global__ static void multiCUDA(const float * a, const float * b, float * c, int n) {
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    // 暂时没有看懂
    const int idx = bid * THREAD_NUM + tid;
    const int row = idx / n;
    const int column = idx % n;

    if (row < n && column < n) {
        float t = 0;
        for (int i = 0; i < n; i++) {
            t += a[row * n + i] * b[i * n + column];
        }

        c[row * n + column] = t;
    }
}

void gpuMatrix(const float * a, const float * b, float * gpuR, int n) {
    float * cuda_a;
    float * cuda_b;
    float * cuda_c;


    hipMalloc((void**)&cuda_a, sizeof(float) * n * n);
    hipMalloc((void**)&cuda_b, sizeof(float) * n * n);
    hipMalloc((void**)&cuda_c, sizeof(float) * n * n);

    hipMemcpy(cuda_a, a, sizeof(float) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(cuda_b, b, sizeof(float) * n * n, hipMemcpyHostToDevice);

    // test time
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    multiCUDA<<< blocks_num, THREAD_NUM, 0 >>>(cuda_a, cuda_b, cuda_c, n);

    hipEventRecord(stop, 0);
    float elapsedTime;

    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    std::cout << "GPU time: " << elapsedTime << std::endl;

    hipMemcpy(gpuR, cuda_c, sizeof(float) * n * n, hipMemcpyDeviceToHost);

    hipFree(cuda_a);
    hipFree(cuda_b);
    hipFree(cuda_c);

}


