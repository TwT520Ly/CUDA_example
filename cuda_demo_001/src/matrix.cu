#include "hip/hip_runtime.h"
#include "matrix.cuh"
#include <iostream>
#include <cstdio>
#include <cstdlib>


__global__ void add(int a, int b, int *c) {
    *c = a + b;
}

int add(int a, int b) {
    int c;
    int *dev_c;
    hipMalloc((void **)&dev_c, sizeof(int));
    add <<<1, 1>>> (a, b, dev_c);
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_c);
    return c;

}

bool initCUDA() {
    int count;

    // 获取cuda数目
    hipGetDeviceCount(&count);

    std::cout << "Cuda number: "<< count << std::endl;

    if (count == 0) {
        fprintf(stderr, "There is no device\n");
        return false;
    }
    int i;

    for (i=0; i<count; i++) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {

            if (prop.major >= 1) {
                break;
            }
        }
    }

    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x. \n");
        return false;
    }

}
