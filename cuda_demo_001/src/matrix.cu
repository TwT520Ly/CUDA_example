#include "hip/hip_runtime.h"
#include "matrix.cuh"
#include <iostream>
#include <cstdio>
#include <cstdlib>


__global__ void add(int a, int b, int *c) {
    *c = a + b;
}

int add(int a, int b) {
    int c;
    int *dev_c;
    hipMalloc((void **)&dev_c, sizeof(int));
    add <<<1, 1>>> (a, b, dev_c);
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_c);
    return c;

}

bool initCUDA() {
    int count;

    // 获取cuda数目
    hipGetDeviceCount(&count);

    std::cout << "Cuda number: "<< count << std::endl;

    if (count == 0) {
        fprintf(stderr, "There is no device\n");
        return false;
    }
    int i;

    for (i=0; i<count; i++) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            // std::cout << prop.multiProcessorCount << std::endl;
            // std::cout << prop.name << std::endl;
            if (prop.major >= 1) {
                break;
            }
        }
    }

    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x. \n");
        return false;
    }
    return true;
}


__global__ static void sumOfSquares(int* num, int* result, int DATA_SIZE) {
    int sum = 0;
    for (int i=0; i<DATA_SIZE; i++) {
        sum += num[i] * num[i] * num[i];
    }

    *result = sum;
}

int sumOfSquares(int* data, int DATA_SIZE) {
    int* gpudata, *result;

    hipMalloc((void**)&gpudata, sizeof(int) * DATA_SIZE);
    hipMalloc((void**)&result, sizeof(int));

    hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice);

    sumOfSquares<<<1, 1, 0>>>(gpudata, result, DATA_SIZE);

    int sum_gpu;
    hipMemcpy(&sum_gpu, result, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(gpudata);
    hipFree(result);

    return sum_gpu;
}