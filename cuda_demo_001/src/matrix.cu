#include "hip/hip_runtime.h"
#include "matrix.cuh"
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#define MAX_THREAD 512
#define MAX_BLOCK 64

__global__ void add(int a, int b, int *c) {
    *c = a + b;
}

int add(int a, int b) {
    int c;
    int *dev_c;
    hipMalloc((void **)&dev_c, sizeof(int));
    add <<<1, 1>>> (a, b, dev_c);
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_c);
    return c;
}

bool initCUDA() {
    int count;

    // 获取cuda数目
    hipGetDeviceCount(&count);

    std::cout << "Cuda number: "<< count << std::endl;

    if (count == 0) {
        fprintf(stderr, "There is no device\n");
        return false;
    }
    int i;

    for (i=0; i<count; i++) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            // std::cout << prop.multiProcessorCount << std::endl;
            std::cout << prop.clockRate << std::endl;
            if (prop.major >= 1) {
                break;
            }
        }
    }

    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x. \n");
        return false;
    }

    hipSetDevice(i);
    return true;
}


__global__ static void sumOfSquares(int* num, int* result, int DATA_SIZE) {
    // 获取线程编号
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    int sum = 0;
    for (int i=bid * MAX_THREAD + tid; i< DATA_SIZE; i += MAX_THREAD * MAX_BLOCK) {
        sum += num[i] * num[i] * num[i];
    }
    result[tid + bid * MAX_THREAD] = sum;
}

int sumOfSquares_gpu(int* data, int DATA_SIZE) {
    int* gpudata;
    int* result;

    hipMalloc((void**)&gpudata, sizeof(int) * DATA_SIZE);
    hipMalloc((void**)&result, sizeof(int) * MAX_THREAD * MAX_BLOCK);

    hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice);

    // test time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    sumOfSquares<<<MAX_BLOCK, MAX_THREAD, 0>>>(gpudata, result, DATA_SIZE);

    int sum_gpu[MAX_THREAD * MAX_BLOCK];

    hipMemcpy(&sum_gpu, result, sizeof(int) * MAX_THREAD * MAX_BLOCK, hipMemcpyDeviceToHost);

    int sum = 0;
    for (int i=0; i<MAX_THREAD * MAX_BLOCK; i++) {
        sum += sum_gpu[i];
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);


    std::cout << "GPU time: " <<  elapsedTime << std::endl;
    hipFree(gpudata);
    hipFree(result);

    return sum;
}

int sumOfSquares_cpu(int* data, int DATA_SIZE) {
    int sum_cpu = 0;
    clock_t start_cpu = clock();
    for (int i=0; i<DATA_SIZE; i++) {
        sum_cpu += data[i] * data[i] * data[i];
    }

    clock_t end_cpu = clock();

    std::cout << "CPU time: " << (double)(end_cpu - start_cpu)* 1000.0 / (CLOCKS_PER_SEC) << std::endl;
    return sum_cpu;
}