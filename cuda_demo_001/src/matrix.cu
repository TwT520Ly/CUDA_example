#include "hip/hip_runtime.h"
#include "matrix.cuh"
#include <iostream>
__global__ void add(int a, int b, int *c) {
    *c = a + b;
}

int add(int a, int b) {
    int c;
    int *dev_c;
    hipMalloc((void **)&dev_c, sizeof(int));
    add <<<1, 1>>> (a, b, dev_c);
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_c);
    return c;

}

bool initCUDA() {
    int count;

    // 获取cuda数目
    hipGetDeviceCount(&count);

    std::cout << "Cuda number: "<< count << std::endl;
}