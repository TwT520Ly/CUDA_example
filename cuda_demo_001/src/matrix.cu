#include "hip/hip_runtime.h"
#include "matrix.cuh"
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <ctime>

__global__ void add(int a, int b, int *c) {
    *c = a + b;
}

int add(int a, int b) {
    int c;
    int *dev_c;
    hipMalloc((void **)&dev_c, sizeof(int));
    add <<<1, 1>>> (a, b, dev_c);
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_c);
    return c;
}

bool initCUDA() {
    int count;

    // 获取cuda数目
    hipGetDeviceCount(&count);

    std::cout << "Cuda number: "<< count << std::endl;

    if (count == 0) {
        fprintf(stderr, "There is no device\n");
        return false;
    }
    int i;

    for (i=0; i<count; i++) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            // std::cout << prop.multiProcessorCount << std::endl;
            std::cout << prop.clockRate << std::endl;
            if (prop.major >= 1) {
                break;
            }
        }
    }

    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x. \n");
        return false;
    }

    hipSetDevice(i);
    return true;
}


__global__ static void sumOfSquares(int* num, int* result, int DATA_SIZE, clock_t* time) {
    int sum = 0;
    clock_t start_gpu = clock();
    for (int i=0; i<DATA_SIZE; i++) {
        sum += num[i] * num[i] * num[i];
    }
    clock_t end_gpu = clock();
    *result = sum;
    *time = end_gpu - start_gpu;
}

int sumOfSquares_gpu(int* data, int DATA_SIZE) {
    int* gpudata;
    int* result;
    clock_t* time;

    hipMalloc((void**)&gpudata, sizeof(int) * DATA_SIZE);
    hipMalloc((void**)&result, sizeof(int));
    hipMalloc((void**)&time, sizeof(clock_t));

    hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice);

    sumOfSquares<<<1, 1, 0>>>(gpudata, result, DATA_SIZE, time);

    int sum_gpu;
    clock_t time_gpu;

    hipMemcpy(&sum_gpu, result, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&time_gpu, time, sizeof(clock_t), hipMemcpyDeviceToHost);
    // clockRate: 1582000 kHZ
    std::cout << "GPU time: " << (double)(time_gpu) / (1582000 * 1000.0) << std::endl;

    hipFree(gpudata);
    hipFree(result);
    hipFree(time);

    return sum_gpu;
}

int sumOfSquares_cpu(int* data, int DATA_SIZE) {
    int sum_cpu = 0;
    clock_t start_cpu = clock();
    for (int i=0; i<DATA_SIZE; i++) {
        sum_cpu += data[i] * data[i] * data[i];
    }

    clock_t end_cpu = clock();

    std::cout << "CPU time: " << (double)(end_cpu - start_cpu) / (CLOCKS_PER_SEC ) << std::endl;
    return sum_cpu;
}