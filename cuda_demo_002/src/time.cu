#include "hip/hip_runtime.h"
#include "time.cuh"
#include <iostream>

__global__ static void time(int* gpudata, int* result, int DATA_SIZE) {
    int sum = 0;
    for (int i=0; i<DATA_SIZE; i++) {
        sum += gpudata[i] * gpudata[i] * gpudata[i];
    }
    *result = sum;
}

void time(int* data, int DATA_SIZE) {
    int* gpudata;
    int* result;

    hipMalloc((void**)&gpudata, sizeof(int) * DATA_SIZE);
    hipMalloc((void**)&result, sizeof(int));

    hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice);

    // test time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    time<<<1, 1, 0>>> (gpudata, result, DATA_SIZE);
    hipEventRecord(stop, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    int sum_result = 0;
    hipMemcpy(&sum_result, result, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Sum result: " << sum_result << std::endl;
    std::cout << "Sum time: " << elapsedTime << std::endl;
    hipFree(result);
    hipFree(gpudata);
}